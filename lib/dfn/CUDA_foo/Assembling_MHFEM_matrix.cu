#include "hip/hip_runtime.h"
#include "Assembling_MHFEM_matrix.cuh"
#include "hip/hip_runtime.h"
#include ""

__global__ void Assemble_on_GPU();

Eigen::SparseMatrix<double> Assembling_MHFEM_matrix(DFN::Mesh_DFN_linear mesh,
                                                    DFN::Domain dom)
{
    //
    Eigen::SparseMatrix<double> K;

    return K;
}